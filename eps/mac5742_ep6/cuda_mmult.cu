#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#ifndef M_PI
#define M_PI 3.14159265
#endif

#include "cuda_mmult_kernels.h"

// define macro OUTPUT to print input & output matrix
#define OUTPUT

// define macro QUERY_DEVICES to print device information
#define QUERY_DEVICES

void checkCUDAError(const char *msg);

void zeroMatrix(float *A, int n);
void dstMatrix(float *A, int n);
void CUDA_freeMatrix(float *Ad);
void printMatrix(char* name, float *A, int n);
void printDeviceInfo(hipDeviceProp_t devProp);
void CUDA_matrixmult_shared_w_prefetch(float *Ad, float* Bd, float* Cd, int n, int repeats);

int main(int argc, char *argv[]) {
   float *A,*B,*C; /* arrays for matrices */
   int n, m; /* n=matrix size, m=repeats */
   
   hipEvent_t start_timer, stop_timer;
   float cpu_time, gpu_time, gpu_time_shared, gpu_time_shared_prefetch;
   
#ifdef QUERY_DEVICES
   // Number of CUDA devices
   int devCount;
   hipGetDeviceCount(&devCount);
   printf("CUDA Device Query...\n");
   printf("There are %d CUDA devices.\n", devCount);

   // Iterate through devices
   for (int i = 0; i < devCount; ++i) 
   {
       // Get device properties
       printf("\nCUDA Device #%d\n", i);
       hipDeviceProp_t devProp;
       hipGetDeviceProperties(&devProp, i);
       printDeviceInfo(devProp);
   }

#endif
  
   if (argc < 3) {
      printf("Error: please specify matrix size and number of multiplications: \n");
      printf("Usage: %s <size> <repeat> \n", argv[0]);
      exit(1);      
   };
   
   /* read matrix size and number of repeats */

   n = atoi(argv[1]);
   m = atoi(argv[2]);

   hipEventCreate(&start_timer);
   hipEventCreate(&stop_timer);

   printf("Matrix mult. of size %d (%d repeats): \n", n, m);

   /* allocate and initialise matrices in host memory */

   int size = n*n*sizeof(float);

   A = (float *) malloc(size);
   dstMatrix(A,n);
   B = (float *) malloc(size);
   dstMatrix(B,n);
   C = (float *) malloc(size);
   zeroMatrix(C,n);
 
   /* allocate matrices in device memory and transfer matrices from host to device memory */
   float *Ad, *Bd, *Cd;
   
   hipMalloc((void**)&Ad,size);
   hipMalloc((void**)&Bd,size);
   hipMalloc((void**)&Cd,size);
   
   hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
   hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
   hipMemcpy(Cd, C, size, hipMemcpyHostToDevice);

   hipMemset(Cd, 0, size);	
   hipEventRecord(start_timer, 0);
   hipEventSynchronize(start_timer);
   CUDA_matrixmult_shared_w_prefetch(Ad, Bd, Cd, n, m);
   hipDeviceSynchronize();
   hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
   hipEventRecord(stop_timer, 0);
   hipEventSynchronize(stop_timer);
   hipEventElapsedTime(&gpu_time_shared_prefetch, start_timer, stop_timer);
   printf("Elapsed GPU time (shared) with prefetching: %.6f s \n", gpu_time_shared_prefetch / 1000.0f);  
 
  /*Find fastest*/

   printf("Performance GPU(shared) with prefetching: %.0f MFlop/s \n", float(m) * (2.0f * n - 1.0f) * n * n / (gpu_time_shared_prefetch / 1000.0f * 1024.f * 1024.f));	

#ifndef OUTPUT
   printMatrix("C", C, n);
#endif
   /* deallocate host matrices, print results */

   free(A);
   free(B);
   free(C);
     
   hipEventDestroy(start_timer);
   hipEventDestroy(stop_timer);

   return(0);
}

/* set Matrix values to zero */
void zeroMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
	    A[i*n+k] = 0;
}

/* initialise Matrix: discrete Sine Transform */
void dstMatrix(float *A, int n)
{
   int i,k;

   for (i=0; i<n; i++)
     for (k=0; k<n; k++)
	    A[i*n+k] = sin( ((i+1)*(k+1)*M_PI)/(n+1));
}

__host__ void CUDA_matrixmult_shared_w_prefetch(float *Ad, float* Bd, float* Cd, int n, int repeats)
{
	int dim_grid_x = (n-1)/TILE_SIZE + 1;
   	int dim_grid_y = (n-1)/TILE_SIZE + 1;
    
   	dim3 dimGrid(dim_grid_x, dim_grid_y);
   	dim3 dimBlock(TILE_SIZE, TILE_SIZE);

   	for(int i = 0 ;i < repeats; i++)
   	{
		matrixMultKernel_overlap<<<dimGrid,dimBlock>>>(Ad, Bd,Cd,n);
	}

  	checkCUDAError("matrix multiplication kernel failed");


}

/* print Matrix */
void printMatrix(char* name, float *A, int n)
{
   int i,k;

   printf("Matrix %s (size %d)\n",name,n);

   for (i=0; i<n; i++) {
     for (k=0; k<n; k++) {
       printf("%.3f ", A[i*n+k]);
     }

     printf("\n");
   }
}

/*
 * helper function to check for errors in CUDA calls
 * source: NVIDIA
 */
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();

    if( hipSuccess != err) {
        fprintf(stderr, "\nCuda error (%s): %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}

#ifdef QUERY_DEVICES
// Print device info
void printDeviceInfo(hipDeviceProp_t devProp) {
    printf("Revision number:               %d.%d\n", devProp.major, devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu MB\n",  devProp.totalGlobalMem / (1024 * 1024));
    printf("Total shared memory per block: %lu kB\n",  devProp.sharedMemPerBlock / 1024);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu MB\n",  devProp.memPitch / (1024 * 1024));
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);

    printf("Maximum dimensions of block:   %d %d %d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    printf("Maximum dimensions of grid:    %d %d %d\n", devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);

    printf("Clock rate:                    %d MHz\n",  devProp.clockRate / 1000);
    printf("Total constant memory:         %lu kB\n",  devProp.totalConstMem / 1024);
    printf("Texture alignment:             %lu B\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    printf("\n");
}
#endif
