
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3
#define N2 N * N

long mnum = 0;

__global__
void minappl(int * arrayM, int * arrayS, int mnum){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = i + 8;
  if (j < N * N * mnum) arrayS[i % 9] = min(arrayM[i], arrayM[j]);
}

int * file2buffer(char * filename) {
	
	FILE * fp = fopen(filename, "r"); if(!fp) {printf("fopen failed at %d", __LINE__); exit(-1);}

	fscanf(fp, "%ld", &mnum);
	int * M = (int *)malloc(mnum * N * N * sizeof(int));

	for(long i = 0; i < mnum; i++){
		fscanf(fp, "%s", stdout);
		for(int l, m, n, o = 0; fscanf(fp, "%d %d %d", &l, &m, &n) == 3; o++){
			M[N * (o + N * i)] = l; 
			M[N * (o + N * i) + 1] = m;
			M[N * (o + N * i) + 2] = n;
		}
	}
	fclose(fp);
	return M;
}

int main(void){
	char filename[] = "sample.txt";
	int * M = file2buffer(filename);
	int S[N2];
	int * dM; hipMalloc(&dM, mnum*N*N*sizeof(int));
	int * dS; hipMalloc(&dS, N*N*sizeof(int));

	hipMemcpy(dM, M, mnum*N*N*sizeof(int), hipMemcpyHostToDevice);
	minappl<<<(N + 255) / 256, 256>>>(dM, dS, mnum);
	hipMemcpy(S, dS, N*N*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N * N; i++)
		printf("S[%d]: %d\n", i, S[i]);

	hipFree(dS);
	hipFree(dM);
	free(M);
}
