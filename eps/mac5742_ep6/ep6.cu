
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 3
#define N2 N * N

long mnum = 0;

__global__
void minappl(int * arrayM, int * arrayS){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	arrayS[i % 9] = min(arrayS[i % 9], arrayM[i]);
}

int * file2buffer(char * filename) {
	
	FILE * fp = fopen(filename, "r"); if(!fp) {printf("fopen failed at %d", __LINE__); exit(-1);}

	fscanf(fp, "%ld", &mnum);
	int * M = (int *)malloc(mnum * N * N * sizeof(int));

	for(long i = 0; i < mnum; i++){
		fscanf(fp, "%s", stdout);
//		printf("stdout: [%s]\n", stdout);
		for(int l, m, n, o = 0; fscanf(fp, "%d %d %d", &l, &m, &n) == 3; o++){
			M[N * (o + N * i)] = l; 
			M[N * (o + N * i) + 1] = m;
			M[N * (o + N * i) + 2] = n;
//			printf("l, m, n: (%d, %d, %d)\n", l, m, n);
		}
	}
	fclose(fp);
	return M;
}

int main(int argc, char * argv[]){

	if(argc != 2) {
		printf("Usage: ./ep6 < filename >\n"); exit(0);
	}
	int * M = file2buffer(argv[1]);
	int S[N2];
	int * dM; hipMalloc(&dM, mnum*N*N*sizeof(int));
	int * dS; hipMalloc(&dS, N*N*sizeof(int));

	/* Initialize S */
	for (int i = 0; i < N * N; i++){
		S[i] = M[i];
//		printf("S[%d][%d]: %d\n", i / N, i % N, S[i]);
	}

	hipMemcpy(dM, M, mnum*N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dS, S, N*N*sizeof(int), hipMemcpyHostToDevice);
	minappl<<<N * N, mnum * N * N>>>(dM, dS);
	hipMemcpy(S, dS, N*N*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N * N; i++)
		printf("S[%d][%d]: %d\n", i / N + 1, i % N + 1, S[i]);

	hipFree(dS);
	hipFree(dM);
	free(M);
}
